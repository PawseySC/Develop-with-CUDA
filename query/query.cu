#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>

void inline check(hipError_t err, const char* filename, int line)
{
	if (err != hipSuccess) 
	{ 
		printf("%s-l%i: %s\n", filename, line, hipGetErrorString(err)); 
		exit(EXIT_FAILURE);
	}
}

int main(int argc, char** argv)
{
	hipError_t err;	// variable for error codes
	int count;			// variable for number of devices
	int device;			// variable for active device id

	err = hipGetDeviceCount(&count);
	check(err, __FILE__, __LINE__);

	printf("\nFound %i devices\n\n", count);

	for (device = 0; device < count; device++)
	{
		err = hipSetDevice(device);
		check(err, __FILE__, __LINE__);

		struct hipDeviceProp_t p;
		err = hipGetDeviceProperties(&p, device);
		check(err, __FILE__, __LINE__);

		printf("Device %i : ", device);
		printf("%s ", p.name);
		printf("with %i SMs\n", p.multiProcessorCount);
	}
	
	printf("\n");

	return EXIT_SUCCESS;
}
